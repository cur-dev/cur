#include <hip/hip_runtime.h>
#include <Rinternals.h>

#include "check.hh"


extern "C" SEXP R_cudaDeviceReset()
{
  CHECK_CUDA(hipDeviceReset());
  return R_NilValue;
}



extern "C" SEXP R_cudaGetDevice()
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 1));
  CHECK_CUDA(hipGetDevice(INTEGER(ret)));
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_cudaGetDeviceCount()
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 1));
  CHECK_CUDA(hipGetDeviceCount(INTEGER(ret)));
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_cudaSetDevice(SEXP device)
{
  CHECK_CUDA(hipSetDevice(INTEGER(device)[0]));
  return R_NilValue;
}
