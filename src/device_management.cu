#include <hip/hip_runtime.h>
#include <Rinternals.h>


extern "C" SEXP R_cudaGetDeviceCount()
{
  SEXP ret;
  hipError_t check;
  
  PROTECT(ret = allocVector(INTSXP, 1));
  check = hipGetDeviceCount(INTEGER(ret));
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_cudaGetDevice()
{
  SEXP ret;
  hipError_t check;
  
  PROTECT(ret = allocVector(INTSXP, 1));
  check = hipGetDevice(INTEGER(ret));
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_cudaSetDevice(SEXP device)
{
  hipError_t check;
  
  check = hipSetDevice(INTEGER(device)[0]);
  return R_NilValue;
}



extern "C" SEXP R_cudaDeviceReset()
{
  hipError_t check;
  
  check = hipDeviceReset();
  return R_NilValue;
}
