#include <hip/hip_runtime.h>
#include <Rinternals.h>


#define COPY_TO_HOST 1
#define COPY_TO_DEVICE 2

#define TYPE_INT 1
#define TYPE_FLOAT 2
#define TYPE_DOUBLE 3

#define newRptr(ptr,Rptr,fin) PROTECT(Rptr = R_MakeExternalPtr(ptr, R_NilValue, R_NilValue));R_RegisterCFinalizerEx(Rptr, fin, TRUE)
#define getRptr(ptr) R_ExternalPtrAddr(ptr)


static inline void cuda_object_finalizer(SEXP Rptr)
{
  hipError_t check;
  void *x = getRptr(Rptr);
  
  if (x == NULL)
    return;
  
  check = hipFree(x);
  R_ClearExternalPtr(Rptr);
}



extern "C" SEXP R_cudaMalloc(SEXP n, SEXP size)
{
  hipError_t check;
  SEXP ret;
  void *x;
  
  size_t len = (size_t) REAL(n)[0] * INTEGER(size)[0];
  check = hipMalloc(&x, len);
  newRptr(x, ret, cuda_object_finalizer);
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_cudaFree(SEXP x_ptr)
{
  hipError_t check;
  void *x = getRptr(x_ptr);
  
  check = hipFree(x);
  return R_NilValue;
}



extern "C" SEXP R_cudaMemGetInfo()
{
  SEXP ret, ret_names;
  SEXP free, total;
  hipError_t check;
  size_t mem_free, mem_total;
  
  PROTECT(ret = allocVector(VECSXP, 2));
  PROTECT(ret_names = allocVector(STRSXP, 2));
  
  PROTECT(free = allocVector(REALSXP, 1));
  PROTECT(total = allocVector(REALSXP, 1));
  
  check = hipMemGetInfo(&mem_free, &mem_total);
  
  REAL(free)[0] = (double) mem_free;
  REAL(total)[0] = (double) mem_total;
  
  SET_VECTOR_ELT(ret, 0, free);
  SET_VECTOR_ELT(ret, 1, total);
  SET_STRING_ELT(ret_names, 0, mkChar("free"));
  SET_STRING_ELT(ret_names, 1, mkChar("total"));
  setAttrib(ret, R_NamesSymbol, ret_names);
  
  UNPROTECT(4);
  return ret;
}



#define SET_ROBJ_PTR(ptr, R_ptr) \
  if (TYPEOF(R_ptr) == INTSXP){ \
    ptr = (void*) INTEGER(R_ptr); \
  } else if (TYPEOF(R_ptr) == REALSXP){ \
    ptr = (void*) REAL(R_ptr); \
  }

extern "C" SEXP R_cudaMemcpy(SEXP dst_, SEXP src_, SEXP count, SEXP size, SEXP kind_)
{
  hipError_t check;
  void *dst;
  void *src;
  
  int kind = INTEGER(kind_)[0];
  size_t len = (size_t) REAL(count)[0] * INTEGER(size)[0];
  
  if (kind == COPY_TO_HOST)
  {
    SET_ROBJ_PTR(dst, dst_);
    src = getRptr(src_);
    check = hipMemcpy(dst, src, len, hipMemcpyDeviceToHost);
  }
  else if (kind == COPY_TO_DEVICE)
  {
    dst = getRptr(dst_);
    SET_ROBJ_PTR(src, src_);
    check = hipMemcpy(dst, src, len, hipMemcpyHostToDevice);
  }
  
  return R_NilValue;
}



extern "C" SEXP R_cudaMemset(SEXP x_ptr, SEXP value, SEXP count, SEXP size)
{
  hipError_t check;
  
  void *x = getRptr(x_ptr);
  size_t len = (size_t) REAL(count)[0] * INTEGER(size)[0];
  check = hipMemset(x, INTEGER(value)[0], len);
  
  return R_NilValue;
}
